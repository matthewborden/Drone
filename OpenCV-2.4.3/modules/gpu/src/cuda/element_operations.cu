#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device
{
    //////////////////////////////////////////////////////////////////////////
    // add

    template <typename T, typename D> struct Add : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a + b);
        }
    };

    template <> struct TransformFunctorTraits< Add<ushort, ushort> > : DefaultTransformFunctorTraits< Add<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<short, short> > : DefaultTransformFunctorTraits< Add<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<int, int> > : DefaultTransformFunctorTraits< Add<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Add<float, float> > : DefaultTransformFunctorTraits< Add<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void add_gpu(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream)
    {
        if (mask.data)
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, Add<T, D>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, Add<T, D>(), WithOutMask(), stream);
    }

    template void add_gpu<uchar, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<uchar, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<schar, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<ushort, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<short, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<int, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<float, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<double, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<double, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    template <typename T, typename D> struct AddScalar : unary_function<T, D>
    {
        AddScalar(double val_) : val(val_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a + val);
        }
        const double val;
    };

    template <> struct TransformFunctorTraits< AddScalar<ushort, ushort> > : DefaultTransformFunctorTraits< AddScalar<ushort, ushort>  >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<short, short> > : DefaultTransformFunctorTraits< AddScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<int, int> > : DefaultTransformFunctorTraits< AddScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddScalar<float, float> > : DefaultTransformFunctorTraits< AddScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void add_gpu(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        AddScalar<T, D> op(val);
        if (mask.data)
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, SingleMask(mask), stream);
        else
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void add_gpu<uchar, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<uchar, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<uchar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<schar, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<schar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<ushort, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<ushort, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<ushort, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<short, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<short, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<short, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<int, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<int, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<int, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<float, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<float, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<float, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void add_gpu<double, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void add_gpu<double, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void add_gpu<double, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // subtract

    template <typename T, typename D> struct Subtract : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a - b);
        }
    };

    template <> struct TransformFunctorTraits< Subtract<ushort, ushort> > : DefaultTransformFunctorTraits< Subtract<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<short, short> > : DefaultTransformFunctorTraits< Subtract<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<int, int> > : DefaultTransformFunctorTraits< Subtract<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Subtract<float, float> > : DefaultTransformFunctorTraits< Subtract<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void subtract_gpu(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream)
    {
        if (mask.data)
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, Subtract<T, D>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, Subtract<T, D>(), WithOutMask(), stream);
    }

    template void subtract_gpu<uchar, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<uchar, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<schar, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<ushort, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<short, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<int, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<float, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<double, uchar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, schar>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, short>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, int>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, float>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<double, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    template <typename T, typename D> struct SubtractScalar : unary_function<T, D>
    {
        SubtractScalar(double val_) : val(val_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a - val);
        }
        const double val;
    };

    template <> struct TransformFunctorTraits< SubtractScalar<ushort, ushort> > : DefaultTransformFunctorTraits< SubtractScalar<ushort, ushort>  >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<short, short> > : DefaultTransformFunctorTraits< SubtractScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<int, int> > : DefaultTransformFunctorTraits< SubtractScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< SubtractScalar<float, float> > : DefaultTransformFunctorTraits< SubtractScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void subtract_gpu(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        SubtractScalar<T, D> op(val);
        if (mask.data)
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, SingleMask(mask), stream);
        else
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void subtract_gpu<uchar, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<uchar, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<uchar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<schar, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<schar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<ushort, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<ushort, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<ushort, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<short, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<short, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<short, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<int, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<int, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<int, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<float, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<float, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<float, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //template void subtract_gpu<double, uchar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, schar>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, short>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, int>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    //template void subtract_gpu<double, float>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);
    template void subtract_gpu<double, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, const PtrStepb& mask, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // multiply

    struct multiply_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
            res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
            res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
            res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);

            return res;
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(multiply_8uc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const PtrStepSz<uchar4>& src1, const PtrStepSzf& src2, const PtrStepSz<uchar4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< PtrStepSz<uint> >(src1), src2, static_cast< PtrStepSz<uint> >(dst), multiply_8uc4_32f(), WithOutMask(), stream);
    }

    struct multiply_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x * b), saturate_cast<short>(a.y * b),
                               saturate_cast<short>(a.z * b), saturate_cast<short>(a.w * b));
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(multiply_16sc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const PtrStepSz<short4>& src1, const PtrStepSzf& src2, const PtrStepSz<short4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< PtrStepSz<short4> >(src1), src2, static_cast< PtrStepSz<short4> >(dst), multiply_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename D> struct Multiply : binary_function<T, T, D>
    {
        Multiply(float scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(scale * a * b);
        }
        const float scale;
    };
    template <typename T> struct Multiply<T, double> : binary_function<T, T, double>
    {
        Multiply(double scale_) : scale(scale_) {}
        __device__ __forceinline__ double operator ()(T a, T b) const
        {
            return scale * a * b;
        }
        const double scale;
    };
    template <> struct Multiply<int, int> : binary_function<int, int, int>
    {
        Multiply(double scale_) : scale(scale_) {}
        __device__ __forceinline__ int operator ()(int a, int b) const
        {
            return saturate_cast<int>(scale * a * b);
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Multiply<ushort, ushort> > : DefaultTransformFunctorTraits< Multiply<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<short, short> > : DefaultTransformFunctorTraits< Multiply<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<int, int> > : DefaultTransformFunctorTraits< Multiply<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Multiply<float, float> > : DefaultTransformFunctorTraits< Multiply<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> struct MultiplyCaller
    {
        static void call(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream)
        {
            Multiply<T, D> op(static_cast<float>(scale));
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
        }
    };
    template <typename T> struct MultiplyCaller<T, double>
    {
        static void call(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&scale) );
            Multiply<T, double> op(scale);
            cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<double>)dst, op, WithOutMask(), stream);
        }
    };
    template <> struct MultiplyCaller<int, int>
    {
        static void call(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&scale) );
            Multiply<int, int> op(scale);
            cv::gpu::device::transform((PtrStepSz<int>)src1, (PtrStepSz<int>)src2, (PtrStepSz<int>)dst, op, WithOutMask(), stream);
        }
    };

    template <typename T, typename D> void multiply_gpu(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream)
    {
        MultiplyCaller<T, D>::call(src1, src2, dst, scale, stream);
    }

    template void multiply_gpu<uchar, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<uchar, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<schar, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<ushort, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<short, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<int, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<float, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<double, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<double, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct MultiplyScalar : unary_function<T, D>
    {
        MultiplyScalar(double val_, double scale_) : val(val_), scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(scale * a * val);
        }
        const double val;
        const double scale;
    };

    template <> struct TransformFunctorTraits< MultiplyScalar<ushort, ushort> > : DefaultTransformFunctorTraits< MultiplyScalar<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<short, short> > : DefaultTransformFunctorTraits< MultiplyScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<int, int> > : DefaultTransformFunctorTraits< MultiplyScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< MultiplyScalar<float, float> > : DefaultTransformFunctorTraits< MultiplyScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void multiply_gpu(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        MultiplyScalar<T, D> op(val, scale);
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void multiply_gpu<uchar, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<uchar, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<uchar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<schar, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<schar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<ushort, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<ushort, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<ushort, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<short, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<short, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<short, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<int, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<int, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<int, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<float, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<float, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<float, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void multiply_gpu<double, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void multiply_gpu<double, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void multiply_gpu<double, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // divide

    struct divide_8uc4_32f : binary_function<uchar4, float, uchar4>
    {
        __device__ __forceinline__ uchar4 operator ()(uchar4 a, float b) const
        {
            return b != 0 ? make_uchar4(saturate_cast<uchar>(a.x / b), saturate_cast<uchar>(a.y / b),
                                        saturate_cast<uchar>(a.z / b), saturate_cast<uchar>(a.w / b))
                          : make_uchar4(0,0,0,0);
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(divide_8uc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const PtrStepSz<uchar4>& src1, const PtrStepSzf& src2, const PtrStepSz<uchar4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< PtrStepSz<uchar4> >(src1), src2, static_cast< PtrStepSz<uchar4> >(dst), divide_8uc4_32f(), WithOutMask(), stream);
    }


    struct divide_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return b != 0 ? make_short4(saturate_cast<short>(a.x / b), saturate_cast<short>(a.y / b),
                                        saturate_cast<short>(a.z / b), saturate_cast<short>(a.w / b))
                          : make_short4(0,0,0,0);
        }
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(divide_16sc4_32f)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const PtrStepSz<short4>& src1, const PtrStepSzf& src2, const PtrStepSz<short4>& dst, hipStream_t stream)
    {
        cv::gpu::device::transform(static_cast< PtrStepSz<short4> >(src1), src2, static_cast< PtrStepSz<short4> >(dst), divide_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename D> struct Divide : binary_function<T, T, D>
    {
        Divide(double scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(a * scale / b) : 0;
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Divide<ushort, ushort> > : DefaultTransformFunctorTraits< Divide<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<short, short> > : DefaultTransformFunctorTraits< Divide<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<int, int> > : DefaultTransformFunctorTraits< Divide<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Divide<float, float> > : DefaultTransformFunctorTraits< Divide<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        Divide<T, D> op(scale);
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<short, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<int, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<float, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<double, uchar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, schar >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, ushort>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, short >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, int   >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, float >(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<double, double>(const PtrStepSzb& src1, const PtrStepSzb& src2, const PtrStepSzb& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct DivideScalar : unary_function<T, D>
    {
        DivideScalar(double val_, double scale_) : val(val_), scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(scale * a / val);
        }
        const double val;
        const double scale;
    };

    template <> struct TransformFunctorTraits< DivideScalar<ushort, ushort> > : DefaultTransformFunctorTraits< DivideScalar<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<short, short> > : DefaultTransformFunctorTraits< DivideScalar<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<int, int> > : DefaultTransformFunctorTraits< DivideScalar<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< DivideScalar<float, float> > : DefaultTransformFunctorTraits< DivideScalar<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        cudaSafeCall( cudaSetDoubleForDevice(&scale) );
        DivideScalar<T, D> op(val, scale);
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<uchar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<schar, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<ushort, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<ushort, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<short, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<short, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<short, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<int, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<int, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<int, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<float, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<float, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<float, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    //template void divide_gpu<double, uchar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, schar >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, ushort>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, short >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, int   >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    //template void divide_gpu<double, float >(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);
    template void divide_gpu<double, double>(const PtrStepSzb& src1, double val, const PtrStepSzb& dst, double scale, hipStream_t stream);

    template <typename T, typename D> struct Reciprocal : unary_function<T, D>
    {
        Reciprocal(double scale_) : scale(scale_) {}
        __device__ __forceinline__ D operator ()(T a) const
        {
            return a != 0 ? saturate_cast<D>(scale / a) : 0;
        }
        const double scale;
    };

    template <> struct TransformFunctorTraits< Reciprocal<ushort, ushort> > : DefaultTransformFunctorTraits< Reciprocal<ushort, ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<short, short> > : DefaultTransformFunctorTraits< Reciprocal<short, short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<int, int> > : DefaultTransformFunctorTraits< Reciprocal<int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Reciprocal<float, float> > : DefaultTransformFunctorTraits< Reciprocal<float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T, typename D> void divide_gpu(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&scalar) );
        Reciprocal<T, D> op(scalar);
        cv::gpu::device::transform((PtrStepSz<T>)src2, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

    template void divide_gpu<uchar, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<uchar, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<uchar, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<uchar, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<uchar, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<uchar, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<uchar, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<schar, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<schar, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<ushort, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<ushort, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<ushort, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<ushort, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<ushort, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<ushort, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<ushort, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<short, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<short, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<short, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<short, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<short, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<short, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<short, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<int, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<int, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<int, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<int, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<int, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<int, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<int, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<float, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<float, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<float, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<float, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<float, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<float, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<float, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //template void divide_gpu<double, uchar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<double, schar >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<double, ushort>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<double, short >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<double, int   >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    //template void divide_gpu<double, float >(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);
    template void divide_gpu<double, double>(double scalar, const PtrStepSzb& src2, const PtrStepSzb& dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // absdiff

    template <typename T> struct Absdiff : binary_function<T, T, T>
    {
        static __device__ __forceinline__ int abs(int a)
        {
            return ::abs(a);
        }
        static __device__ __forceinline__ float abs(float a)
        {
            return ::fabsf(a);
        }
        static __device__ __forceinline__ double abs(double a)
        {
            return ::fabs(a);
        }

        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return saturate_cast<T>(::abs(a - b));
        }
    };

    template <> struct TransformFunctorTraits< Absdiff<ushort> > : DefaultTransformFunctorTraits< Absdiff<ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<short> > : DefaultTransformFunctorTraits< Absdiff<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<int> > : DefaultTransformFunctorTraits< Absdiff<int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< Absdiff<float> > : DefaultTransformFunctorTraits< Absdiff<float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void absdiff_gpu(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<T>)dst, Absdiff<T>(), WithOutMask(), stream);
    }

    //template void absdiff_gpu<uchar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<schar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    //template void absdiff_gpu<ushort>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<short >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<int   >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    //template void absdiff_gpu<float >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<double>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template <typename T> struct AbsdiffScalar : unary_function<T, T>
    {
        AbsdiffScalar(double val_) : val(val_) {}
        __device__ __forceinline__ T operator ()(T a) const
        {
            return saturate_cast<T>(::fabs(a - val));
        }
        double val;
    };

    template <> struct TransformFunctorTraits< AbsdiffScalar<ushort> > : DefaultTransformFunctorTraits< AbsdiffScalar<ushort> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<short> > : DefaultTransformFunctorTraits< AbsdiffScalar<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<int> > : DefaultTransformFunctorTraits< AbsdiffScalar<int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AbsdiffScalar<float> > : DefaultTransformFunctorTraits< AbsdiffScalar<float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void absdiff_gpu(const PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&val) );
        AbsdiffScalar<T> op(val);
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)dst, op, WithOutMask(), stream);
    }

    //template void absdiff_gpu<uchar >(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<schar >(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    //template void absdiff_gpu<ushort>(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<short >(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<int   >(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    //template void absdiff_gpu<float >(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absdiff_gpu<double>(const PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

    template <template <typename> class Op, typename T>
    struct Compare: binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T src1, T src2) const
        {
            Op<T> op;
            return static_cast<uchar>(static_cast<int>(op(src1, src2)) * 255);
        }
    };

#define IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(op, type, block_dim_y, shift) \
    template <> struct TransformFunctorTraits< Compare<op, type> > : DefaultTransformFunctorTraits< Compare<op, type> > \
    { \
        enum { smart_block_dim_y = block_dim_y }; \
        enum { smart_shift = shift }; \
    };

    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, float, 8, 4)

#undef IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS

    template <template <typename> class Op, typename T> void compare(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        Compare<Op, T> op;
        cv::gpu::device::transform(static_cast< PtrStepSz<T> >(src1), static_cast< PtrStepSz<T> >(src2), dst, op, WithOutMask(), stream);
    }

    template <typename T> void compare_eq(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        compare<equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_ne(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        compare<not_equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_lt(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        compare<less, T>(src1, src2, dst, stream);
    }
    template <typename T> void compare_le(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        compare<less_equal, T>(src1, src2, dst, stream);
    }

    template void compare_eq<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void compare_ne<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void compare_lt<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void compare_le<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void compare_le<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

#define TYPE_VEC(type, cn) typename TypeVec<type, cn>::vec_type

    template <template <typename> class Op, typename T, int cn> struct CompareScalar;
    template <template <typename> class Op, typename T>
    struct CompareScalar<Op, T, 1>: unary_function<T, uchar>
    {
        const T val;

        __host__ explicit CompareScalar(T val_) : val(val_) {}

        __device__ __forceinline__ uchar operator()(T src) const
        {
            Op<T> op;
            return static_cast<uchar>(static_cast<int>(op(src, val)) * 255);
        }
    };
    template <template <typename> class Op, typename T>
    struct CompareScalar<Op, T, 2>: unary_function<TYPE_VEC(T, 2), TYPE_VEC(uchar, 2)>
    {
        const TYPE_VEC(T, 2) val;

        __host__ explicit CompareScalar(TYPE_VEC(T, 2) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 2) operator()(const TYPE_VEC(T, 2) & src) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 2)>::make(
                        static_cast<uchar>(static_cast<int>(op(src.x, val.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.y, val.y)) * 255));
        }
    };
    template <template <typename> class Op, typename T>
    struct CompareScalar<Op, T, 3>: unary_function<TYPE_VEC(T, 3), TYPE_VEC(uchar, 3)>
    {
        const TYPE_VEC(T, 3) val;

        __host__ explicit CompareScalar(TYPE_VEC(T, 3) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 3) operator()(const TYPE_VEC(T, 3) & src) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 3)>::make(
                        static_cast<uchar>(static_cast<int>(op(src.x, val.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.y, val.y)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.z, val.z)) * 255));
        }
    };
    template <template <typename> class Op, typename T>
    struct CompareScalar<Op, T, 4>: unary_function<TYPE_VEC(T, 4), TYPE_VEC(uchar, 4)>
    {
        const TYPE_VEC(T, 4) val;

        __host__ explicit CompareScalar(TYPE_VEC(T, 4) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 4) operator()(const TYPE_VEC(T, 4) & src) const
        {
            Op<T> op;
            return VecTraits<TYPE_VEC(uchar, 4)>::make(
                        static_cast<uchar>(static_cast<int>(op(src.x, val.x)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.y, val.y)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.z, val.z)) * 255),
                        static_cast<uchar>(static_cast<int>(op(src.w, val.w)) * 255));
        }
    };

#undef TYPE_VEC

#define IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(op, type, block_dim_y, shift) \
    template <> struct TransformFunctorTraits< CompareScalar<op, type, 1> > : DefaultTransformFunctorTraits< CompareScalar<op, type, 1> > \
    { \
        enum { smart_block_dim_y = block_dim_y }; \
        enum { smart_shift = shift }; \
    };

    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(not_equal_to, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(greater_equal, float, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, int, 8, 4)
    IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS(less_equal, float, 8, 4)

#undef IMPLEMENT_COMPARE_TRANSFORM_FUNCTOR_TRAITS

    template <template <typename> class Op, typename T, int cn> void compare(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type src_t;
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        T sval[] = {static_cast<T>(val[0]), static_cast<T>(val[1]), static_cast<T>(val[2]), static_cast<T>(val[3])};
        src_t val1 = VecTraits<src_t>::make(sval);

        CompareScalar<Op, T, cn> op(val1);

        cv::gpu::device::transform(static_cast< PtrStepSz<src_t> >(src), static_cast< PtrStepSz<dst_t> >(dst), op, WithOutMask(), stream);
    }

    template <typename T> void compare_eq(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<equal_to, T, 1>,
            compare<equal_to, T, 2>,
            compare<equal_to, T, 3>,
            compare<equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_ne(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<not_equal_to, T, 1>,
            compare<not_equal_to, T, 2>,
            compare<not_equal_to, T, 3>,
            compare<not_equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_lt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<less, T, 1>,
            compare<less, T, 2>,
            compare<less, T, 3>,
            compare<less, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_le(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<less_equal, T, 1>,
            compare<less_equal, T, 2>,
            compare<less_equal, T, 3>,
            compare<less_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_gt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<greater, T, 1>,
            compare<greater, T, 2>,
            compare<greater, T, 3>,
            compare<greater, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void compare_ge(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            compare<greater_equal, T, 1>,
            compare<greater_equal, T, 2>,
            compare<greater_equal, T, 3>,
            compare<greater_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }

    template void compare_eq<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_eq<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void compare_ne<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ne<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void compare_lt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_lt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void compare_le<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_le<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void compare_gt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_gt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void compare_ge<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void compare_ge<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    {
        static __device__ __forceinline__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStepb src, PtrStepb dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows)
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStepb src, PtrStepb dst,
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)),
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStepb src,
                                      const PtrStepb mask, PtrStepb dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn])
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStepb src,
                     const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseNotCaller(int rows, int cols, size_t elem_size1, int cn,
                          const PtrStepb src, PtrStepb dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, static_cast<int>(cols * elem_size1 * cn), src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStepb src,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a | b; }
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a & b; }
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    {
        static __device__ __forceinline__ T call(T a, T b) { return a ^ b; }
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStepb src1,
                                       const PtrStepb src2, PtrStepb dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows)
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStepb src1, const PtrStepb src2,
                      PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
            const PtrStepb mask, PtrStepb dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn])
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                        const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseOrCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                         const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                             const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                          const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStepb src1,
                          const PtrStepb src2, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStepb src1, const PtrStepb src2,
                              const PtrStepb mask, PtrStepb dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStepb, const PtrStepb, const PtrStepb, PtrStepb, hipStream_t);

    //////////////////////////////////////////////////////////////////////////
    // min/max

    namespace detail
    {
        template <size_t size, typename F> struct MinMaxTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct MinMaxTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct MinMaxTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< minimum<T> > : detail::MinMaxTraits< sizeof(T), minimum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< maximum<T> > : detail::MinMaxTraits< sizeof(T), maximum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< minimum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< minimum<T> > >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< maximum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< maximum<T> > >
    {
    };

    template <typename T>
    void min_gpu(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<T>)dst, minimum<T>(), WithOutMask(), stream);
    }

    template void min_gpu<uchar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<schar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<ushort>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<short >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<int   >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<float >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<double>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src1, (PtrStepSz<T>)src2, (PtrStepSz<T>)dst, maximum<T>(), WithOutMask(), stream);
    }

    template void max_gpu<uchar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<schar >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<ushort>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<short >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<int   >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<float >(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<double>(const PtrStepSzb src1, const PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const PtrStepSzb src, T val, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<T>)dst, device::bind2nd(minimum<T>(), val), WithOutMask(), stream);
    }

    template void min_gpu<uchar >(const PtrStepSzb src, uchar  val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<schar >(const PtrStepSzb src, schar  val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<ushort>(const PtrStepSzb src, ushort val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<short >(const PtrStepSzb src, short  val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<int   >(const PtrStepSzb src, int    val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<float >(const PtrStepSzb src, float  val, PtrStepSzb dst, hipStream_t stream);
    template void min_gpu<double>(const PtrStepSzb src, double val, PtrStepSzb dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const PtrStepSzb src, T val, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<T>)dst, device::bind2nd(maximum<T>(), val), WithOutMask(), stream);
    }

    template void max_gpu<uchar >(const PtrStepSzb src, uchar  val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<schar >(const PtrStepSzb src, schar  val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<ushort>(const PtrStepSzb src, ushort val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<short >(const PtrStepSzb src, short  val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<int   >(const PtrStepSzb src, int    val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<float >(const PtrStepSzb src, float  val, PtrStepSzb dst, hipStream_t stream);
    template void max_gpu<double>(const PtrStepSzb src, double val, PtrStepSzb dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // threshold

    namespace detail
    {
        template <size_t size, typename F> struct ThresholdTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct ThresholdTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ThresholdTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< thresh_binary_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_binary_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_inv_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_trunc_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_trunc_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_inv_func<T> >
    {
    };

    template <template <typename> class Op, typename T>
    void threshold_caller(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, T thresh, T maxVal, hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        cv::gpu::device::transform(src, dst, op, WithOutMask(), stream);
    }

    template <typename T>
    void threshold_gpu(const PtrStepSzb& src, const PtrStepSzb& dst, T thresh, T maxVal, int type,
        hipStream_t stream)
    {
        typedef void (*caller_t)(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, T thresh, T maxVal, hipStream_t stream);

        static const caller_t callers[] =
        {
            threshold_caller<thresh_binary_func, T>,
            threshold_caller<thresh_binary_inv_func, T>,
            threshold_caller<thresh_trunc_func, T>,
            threshold_caller<thresh_to_zero_func, T>,
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((PtrStepSz<T>)src, (PtrStepSz<T>)dst, thresh, maxVal, stream);
    }

    template void threshold_gpu<uchar>(const PtrStepSzb& src, const PtrStepSzb& dst, uchar thresh, uchar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<schar>(const PtrStepSzb& src, const PtrStepSzb& dst, schar thresh, schar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<ushort>(const PtrStepSzb& src, const PtrStepSzb& dst, ushort thresh, ushort maxVal, int type, hipStream_t stream);
    template void threshold_gpu<short>(const PtrStepSzb& src, const PtrStepSzb& dst, short thresh, short maxVal, int type, hipStream_t stream);
    template void threshold_gpu<int>(const PtrStepSzb& src, const PtrStepSzb& dst, int thresh, int maxVal, int type, hipStream_t stream);
    template void threshold_gpu<float>(const PtrStepSzb& src, const PtrStepSzb& dst, float thresh, float maxVal, int type, hipStream_t stream);
    template void threshold_gpu<double>(const PtrStepSzb& src, const PtrStepSzb& dst, double thresh, double maxVal, int type, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // pow

    template<typename T, bool Signed = device::numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            return saturate_cast<T>(__powf((float)e, power));
        }
    };
    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            T res = saturate_cast<T>(__powf((float)e, power));

            if ((e < 0) && (1 & static_cast<int>(power)))
                res *= -1;

            return res;
        }
    };
    template<> struct PowOp<float> : unary_function<float, float>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ float operator()(float e) const
        {
            return __powf(::fabs(e), power);
        }
    };
    template<> struct PowOp<double> : unary_function<double, double>
    {
        const double power;

        PowOp(double power_) : power(power_) {}

        __device__ __forceinline__ double operator()(double e) const
        {
            return ::pow(::fabs(e), power);
        }
    };

    namespace detail
    {
        template <size_t size, typename T> struct PowOpTraits : DefaultTransformFunctorTraits< PowOp<T> >
        {
        };
        template <typename T> struct PowOpTraits<1, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 8 };
        };
        template <typename T> struct PowOpTraits<2, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_shift = 4 };
        };
        template <typename T> struct PowOpTraits<4, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< PowOp<T> > : detail::PowOpTraits<sizeof(T), T>
    {
    };

    template<typename T>
    void pow_caller(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream)
    {
        cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<T>)dst, PowOp<T>(power), WithOutMask(), stream);
    }

    template void pow_caller<uchar>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<schar>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<short>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<ushort>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<int>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<float>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow_caller<double>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // addWeighted

    namespace detail
    {
        template <typename T> struct UseDouble
        {
            enum {value = 0};
        };
        template <> struct UseDouble<int>
        {
            enum {value = 1};
        };
        template <> struct UseDouble<float>
        {
            enum {value = 1};
        };
        template <> struct UseDouble<double>
        {
            enum {value = 1};
        };
    }
    template <typename T1, typename T2, typename D> struct UseDouble
    {
        enum {value = (detail::UseDouble<T1>::value || detail::UseDouble<T2>::value || detail::UseDouble<D>::value)};
    };

    namespace detail
    {
        template <typename T1, typename T2, typename D, bool useDouble> struct AddWeighted;
        template <typename T1, typename T2, typename D> struct AddWeighted<T1, T2, D, false> : binary_function<T1, T2, D>
        {
            AddWeighted(double alpha_, double beta_, double gamma_) : alpha(static_cast<float>(alpha_)), beta(static_cast<float>(beta_)), gamma(static_cast<float>(gamma_)) {}

            __device__ __forceinline__ D operator ()(T1 a, T2 b) const
            {
                return saturate_cast<D>(a * alpha + b * beta + gamma);
            }

            const float alpha;
            const float beta;
            const float gamma;
        };
        template <typename T1, typename T2, typename D> struct AddWeighted<T1, T2, D, true> : binary_function<T1, T2, D>
        {
            AddWeighted(double alpha_, double beta_, double gamma_) : alpha(alpha_), beta(beta_), gamma(gamma_) {}

            __device__ __forceinline__ D operator ()(T1 a, T2 b) const
            {
                return saturate_cast<D>(a * alpha + b * beta + gamma);
            }

            const double alpha;
            const double beta;
            const double gamma;
        };
    }
    template <typename T1, typename T2, typename D> struct AddWeighted : detail::AddWeighted<T1, T2, D, UseDouble<T1, T2, D>::value>
    {
        AddWeighted(double alpha_, double beta_, double gamma_) : detail::AddWeighted<T1, T2, D, UseDouble<T1, T2, D>::value>(alpha_, beta_, gamma_) {}
    };

    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<short, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, short> > : DefaultTransformFunctorTraits< AddWeighted<short, short, short> >
    {
        enum { smart_shift = 4 };
    };

    template <> struct TransformFunctorTraits< AddWeighted<int, int, int> > : DefaultTransformFunctorTraits< AddWeighted<int, int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, int, float> > : DefaultTransformFunctorTraits< AddWeighted<int, int, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, int> > : DefaultTransformFunctorTraits< AddWeighted<int, float, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, float> > : DefaultTransformFunctorTraits< AddWeighted<int, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, int> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, float> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T1, typename T2, typename D>
    void addWeighted_gpu(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream)
    {
        if (UseDouble<T1, T2, D>::value)
        {
            cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
            cudaSafeCall( cudaSetDoubleForDevice(&beta) );
            cudaSafeCall( cudaSetDoubleForDevice(&gamma) );
        }

        AddWeighted<T1, T2, D> op(alpha, beta, gamma);

        cv::gpu::device::transform(static_cast< PtrStepSz<T1> >(src1), static_cast< PtrStepSz<T2> >(src2), static_cast< PtrStepSz<D> >(dst), op, WithOutMask(), stream);
    }

    template void addWeighted_gpu<uchar, uchar, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, schar, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, ushort, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, short, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, int, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<schar, schar, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, ushort, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, short, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, int, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<ushort, ushort, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, short, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, int, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<short, short, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<short, int, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<short, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<short, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<int, int, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<int, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<int, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<float, float, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);

    template void addWeighted_gpu<float, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);



    template void addWeighted_gpu<double, double, uchar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, schar>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, ushort>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, short>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, int>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, float>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, double>(const PtrStepSzb& src1, double alpha, const PtrStepSzb& src2, double beta, double gamma, const PtrStepSzb& dst, hipStream_t stream);
}}} // namespace cv { namespace gpu { namespace device

#endif /* CUDA_DISABLER */